#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "device_atomic_functions.h"

#include <stdio.h>
#include "Bfs.cuh"

__global__ void bfs_kernel(unsigned int* current_set, unsigned int* new_set,
	int current_set_size, int* current_set_size_new,
	Node* node_list, Edge* edge_list, int* color, int* cost, int level)
{

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j = tid; j<current_set_size; j += blockDim.x*gridDim.x) {
		unsigned int index = current_set[j];// fetch one from the current set
		current_set[j] = 0;                 // erase it
		cost[index] = level;
		Node cur_node = node_list[index];
		for (int i = cur_node.start; i < cur_node.start + cur_node.edge_num; i++)
		{
			unsigned int id = edge_list[i].dest;
			int its_color = atomicExch((int*)&color[id], BLACK);
			if (its_color == WHITE) {
				int write_position = atomicAdd((int*) &(*current_set_size_new), 1);
				new_set[write_position] = id;
			}
		}
	}
}

void callBFSKernel(const unsigned int blocks,
	const unsigned int threadsPerBlock,
	unsigned int* current_set, unsigned int* new_set,
	int current_set_size, int* current_set_size_new,
	Node* node_list, Edge* edge_list, int* color, int* cost, int level)
{
	bfs_kernel << <blocks, threadsPerBlock >> > (current_set, new_set, current_set_size,
		current_set_size_new, node_list, edge_list, color, cost, level);
}

